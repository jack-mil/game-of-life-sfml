#include "hip/hip_runtime.h"
/*
Author: Jackson Miller
Class: ECE6122 A
Last Date Modified: 2024-10-10

Description:
Implementation of Game of Life rules. Does not display the simulated world.
*/

#include <random>

#include <hip/hip_runtime.h>

#include "Life.cuh"
#include "Mode.hpp"
#include "helper_cuda.cuh"
#include <cassert>

/** Only constructor for Life class */
Life::Life(size_t height, size_t width, Mode mode, uint threads)
    : m_height{height},                           // height in cells
      m_width{width},                             // width in cells
      m_mode{mode},                               // Mode enum specifies the Cuda memory copy technique
      m_threads{nullptr},                         // Dim of threads per execution block
      m_blocks{nullptr},                          // Dim of blocks in the grid (round up)
      m_bfr_current(height * width, State::Dead), // allocate grid buffers with dead cells
      m_bfr_next(height * width, State::Dead)     // second buffer
{

    int devID = findCudaDevice();

    hipDeviceProp_t props;
    checkCudaErrors(hipGetDeviceProperties(&props, devID));

    if (!props.managedMemory) {
        // Game of life requires being run on a device that supports Unified Memory
        fprintf(stderr, "Unified Memory not supported on this device\n");
        exit(EXIT_WAIVED);
    }

    if (!threads > props.maxThreadsPerBlock) {
        fprintf(stderr, "Threads per block cannot be greater than: %d", props.maxThreadsPerBlock);
        exit(EXIT_WAIVED);
    }
    printDeviceStats(props);

    assert(mode == Mode::Normal);

    // Allocte device memory (host already allocated as a vector<State>)
    checkCudaErrors(hipMallocPitch(&d_bfr_current, &d_current_pitch,
                                    sizeof(State) * m_width, m_height));
    checkCudaErrors(hipMallocPitch(&d_bfr_next, &d_next_pitch,
                                    sizeof(State) * m_width, m_height));

    // assume threads is multiple of 32
    const uint long_side = threads >= 512   ? 32
                           : threads >= 128 ? 16
                                            : 8;

    auto y_dim = threads / long_side;
    auto x_dim = threads / y_dim;
    m_threads  = new dim3{x_dim, y_dim}; // AKA block size
    // need enough blocks for a thread for every cell (round up)
    m_blocks = new dim3{
        ((uint)m_width + m_threads->x - 1) / m_threads->x,
        ((uint)m_height + m_threads->y - 1) / m_threads->y,
    }; // AKA Grid size

    printf("Life %dx%d\n", m_width, m_height);
    printf("CUDA : blockDim=(%u,%u,%u), gridDim=(%u,%u,%u)\n",
           m_threads->x, m_threads->y, m_threads->z,
           m_blocks->x, m_blocks->y, m_blocks->z);

    // Seed the host starting universe with a random state
    this->seedRandom();
}

/** Destructor frees allocated memory */
Life::~Life()
{
    // free heap data
    delete m_threads;
    delete m_blocks;
    // free Cuda device memory
    checkCudaErrors(hipFree(d_bfr_current));
    checkCudaErrors(hipFree(d_bfr_next));
}

/**
 * Initialize each cell of the grid with a random true/false value
 * Makes use of the bernoulli distribution and hardware random number generator.
 * */
void Life::seedRandom()
{
    std::default_random_engine  gen(std::random_device{}());
    std::bernoulli_distribution coin_flip(0.5); // uniform boolean true/false distribution
    // for (size_t i = 0; i < m_width * m_height; i++) {
    //     m_bfr_current[i] = static_cast<State>(coin_flip(gen));
    // }
    for (auto& cell : m_bfr_current) {
        cell = static_cast<State>(coin_flip(gen));
    }
}

/**
 * Convert row,col specifier to the 1D vector access
 * Reads from current state
 */
inline Life::State Life::getCell(size_t row, size_t col) const
{
    return m_bfr_current[row * m_width + col];
}

/**
 * Return a collection of all the row, col points that are currently alive.
 * Compiler will use RVO and move semantics so this avoids unnecessary copying (probably)
 */
std::vector<std::pair<int, int>> Life::getLiveCells() const
{
    std::vector<std::pair<int, int>> liveCells;
    for (size_t row = 0; row < m_height; ++row) {
        for (size_t col = 0; col < m_width; ++col) {
            if (this->getCell(row, col) == State::Alive) {
                liveCells.emplace_back(row, col);
            }
        }
    }
    return liveCells;
}
/**
 * Run one iteration of the Game of Life
 * Use different parallelization techniques according to the current mode
 */
void Life::doOneGeneration()
{
    // Pick an implementation
    switch (m_mode) {
    case Mode::Normal:
        this->updateCudaNormal();
        break;
    case Mode::Managed:
        this->updateCudaManaged();
        break;
    case Mode::Pinned:
        this->updateCudaPinned();
        break;
    }

    // swap the std::vectors. This only swaps the underlying pointers,
    // not the contained data. Very cheap and fast (hopefully)
    std::swap(m_bfr_current, m_bfr_next);
}

/* Device kernel using a 2d grid and block size */
__global__ void deviceOneGeneration(uint8_t* now, uint8_t* next,
                                    size_t pitch_now, size_t pitch_next,
                                    int width, int height)
{
    // "2d" arrangement of threads and blocks
    uint x_index = (blockIdx.x * blockDim.x) + threadIdx.x;
    uint y_index = (blockIdx.y * blockDim.y) + threadIdx.y;

    // printf("px=%ud,%ud\n", x_index, y_index);
    // printf("Tidx=%d, Tidy=%d, Tidz=%d\n", threadIdx.x, threadIdx.y, threadIdx.z);
    // printf("blkX=%d, blkY=%d, blkZ=%d\n", blockDim.x, blockDim.y, blockDim.z);
    // printf("gridX=%d, gridY=%d, gridZ=%d\n", gridDim.x, gridDim.y, gridDim.z);

    /* total number of spawned threads in x direction*/
    uint stride_col = blockDim.x * gridDim.x;
    /* total number of spawned threads in y direction*/
    uint stride_row = blockDim.y * gridDim.y;

    // if (x_index == 0 || x_index >= width) {
    //     return;
    // }
    // if (y_index == 0 || y_index >= height) {
    //     return;
    // }

    // 2d stride loop. Only executed more than once if not enough blocks (unlikely)
    for (size_t y = y_index; y < height; y += stride_row) {
        for (size_t x = x_index; x < width; x += stride_col) {
            // T* pElement = (T*)((char*)BaseAddress + Row * pitch) + Column;

            uint8_t* top_row = now + ((y - 1) * pitch_now);
            uint8_t* mid_row = now + (y * pitch_now);
            uint8_t* bot_row = now + ((y + 1) * pitch_now);

            // sum of whole 9x9 area
            int count = top_row[x - 1] + top_row[x] + top_row[x + 1] +
                        mid_row[x - 1] + mid_row[x] + mid_row[x + 1] +
                        bot_row[x - 1] + bot_row[x] + bot_row[x + 1];

            // uint8_t* next_cell = (uint8_t*)next + (y * pitch_next) + x;

            next[(y * pitch_next) + x] = (count == 3)   ? 1u         // alive
                                         : (count == 4) ? mid_row[x] // no change
                                                        : 0u;        // dies
                                                                     // next[(y * pitch_next) + x] = (count == 3 || (count == 2 && mid_row[x])) ? 1u : 0u;
        }
    }
    // size_t ix = blockDim.x * blockIdx.x + threadIdx.x + 1;
    // size_t iy = blockDim.y * blockIdx.y + threadIdx.y + 1;
    // size_t id = iy * (width + 2) + ix;
}

void Life::updateCudaNormal()
{
    // copy host array (vector) to device
    checkCudaErrors(hipMemcpy2D(d_bfr_current, d_current_pitch,
                                 m_bfr_current.data(), sizeof(State) * m_width, // host pitch is width
                                 sizeof(State) * m_width, m_height,
                                 hipMemcpyHostToDevice));
    // execute kernel
    deviceOneGeneration<<<*m_blocks, *m_threads>>>((uint8_t*)d_bfr_current, (uint8_t*)d_bfr_next,
                                                   d_current_pitch, d_next_pitch,
                                                   m_width, m_height);
    checkCudaErrors(hipPeekAtLastError()); // detect errors in kernel execution
    // copy memory back to device (blocks until kernel done)
    checkCudaErrors(hipMemcpy2D(m_bfr_next.data(), sizeof(State) * m_width, /* pitch==width on host */
                                 d_bfr_next, d_next_pitch,
                                 sizeof(State) * m_width, m_height,
                                 hipMemcpyDeviceToHost));
    // new gen stored in bfr_next
}
void Life::updateCudaManaged()
{
}
void Life::updateCudaPinned()
{
}
