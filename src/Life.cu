#include "hip/hip_runtime.h"
/*
Author: Jackson Miller
Class: ECE6122 A
Last Date Modified: 2024-10-10

Description:
Implementation of Game of Life rules. Does not display the simulated world.
*/

#include <random>

#include <hip/hip_runtime.h>

#include "Life.cuh"
#include "Mode.hpp"
#include "helper_cuda.cuh"
#include <cassert>

size_t d_current_pitch; // width in bytes of allocation
size_t d_next_pitch;    // width in bytes of allocation

/** Only constructor for Life class */
Life::Life(size_t rows, size_t cols, Mode mode, uint threads)
    : m_height{rows},                                    // height in cells
      m_width{cols},                                     // width in cells
      m_mode{mode},                                      // Mode enum specifies the Cuda memory copy technique
      m_threads{threads},                                // Number of threads per execution block
      m_blocks{((rows * cols) + threads - 1) / threads}, // Number of blocks in the grid (round up)
      m_bfr_current(rows * cols, State::Dead),           // allocate grid buffers with dead cells
      m_bfr_next(rows * cols, State::Dead)               // second buffer
{

    int devID = findCudaDevice();

    hipDeviceProp_t deviceProp;
    checkCudaErrors(hipGetDeviceProperties(&deviceProp, devID));

    if (!deviceProp.managedMemory) {
        // Game of life requires being run on a device that supports Unified Memory
        fprintf(stderr, "Unified Memory not supported on this device\n");
        exit(EXIT_WAIVED);
    }

    // Statistics about the GPU device
    printf(
        "> GPU device has %d Multi-Processors, SM %d.%d compute capabilities\n\n",
        deviceProp.multiProcessorCount, deviceProp.major, deviceProp.minor);

    assert(mode == Mode::Normal);

    // Start with a random initial state
    this->seedRandom();

    // Allocte device memory (host already allocated as a vector<State>)
    // checkCudaErrors(hipMalloc(&d_bfr_current, sizeof(State) * m_width * m_height));
    // checkCudaErrors(hipMalloc(&d_bfr_next, sizeof(State) * m_height * m_width));

    checkCudaErrors(hipMallocPitch(&d_bfr_current, &d_current_pitch,
                                    sizeof(State) * m_width, m_height));
    checkCudaErrors(hipMallocPitch(&d_bfr_next, &d_next_pitch,
                                    sizeof(State) * m_width, m_height));
    // checkCudaErrors(hipMalloc(&d_bfr_next, sizeof(State) * m_height * m_width));

    // calculate thread and block size
    // m_block_size = new dim3{threads, threads, 1};
    // uint linGrid = (uint)ceil(m_width + 2 / (float)threads);
    // m_grid_size  = new dim3{linGrid, linGrid, 1};
}

/** Destructor frees allocated memory */
Life::~Life()
{ // free heap data
    // delete m_grid_size;
    // delete m_block_size;

    // free Cuda device memory
    checkCudaErrors(hipFree(d_bfr_current));
    checkCudaErrors(hipFree(d_bfr_next));
}

/**
 * Initialize each cell of the grid with a random true/false value
 * Makes use of the bernoulli distribution and hardware random number generator.
 * */
void Life::seedRandom()
{
    std::default_random_engine  gen(std::random_device{}());
    std::bernoulli_distribution coin_flip(0.5); // uniform boolean true/false distribution
    for (auto& cell : m_bfr_current) {
        cell = static_cast<State>(coin_flip(gen));
    }
}

/**
 * Run one iteration of the Game of Life
 * Use different parallelization techniques according to the current mode
 */
void Life::doOneGeneration()
{
    // Pick an implementation
    switch (m_mode) {
    case Mode::Normal:
        this->updateCudaNormal();
        break;
    case Mode::Managed:
        this->updateCudaManaged();
        break;
    case Mode::Pinned:
        this->updateCudaPinned();
        break;
    }

    // swap the std::vectors. This only swaps the underlying pointers,
    // not the contained data. Very cheap and fast (hopefully)
    // std::swap(m_bfr_current, m_bfr_next);
}

/**
 * Return a collection of all the row, col points that are currently alive.
 * Compiler will use RVO and move semantics so this avoids unnecessary copying (probably)
 */
std::vector<std::pair<int, int>> Life::getLiveCells() const
{
    std::vector<std::pair<int, int>> liveCells;
    for (size_t row = 0; row < m_height; ++row) {
        for (size_t col = 0; col < m_width; ++col) {
            if (this->getCell(row, col) == State::Alive) {
                liveCells.emplace_back(row, col);
            }
        }
    }
    return liveCells;
}

__global__ void deviceOneGeneration(const uint8_t* now, uint8_t* next,
                                    size_t pitch_now, size_t pitch_next,
                                    int width, int height)
{
    // "2d" arrangement of threads and blocks
    int x_index = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y_index = (blockIdx.y * blockDim.y) + threadIdx.y;

    // printf("Tidx=%d, Tidy=%d, Tidz=%d\n", threadIdx.x, threadIdx.y, threadIdx.z);
    // printf("blkX=%d, blkY=%d, blkZ=%d\n", blockDim.x, blockDim.y, blockDim.z);
    // printf("gridX=%d, gridY=%d, gridZ=%d\n", gridDim.x, gridDim.y, gridDim.z);

    /* total number of spawned threads in x direction*/
    int stride_col = blockDim.x * gridDim.x;
    /* total number of spawned threads in y direction*/
    int stride_row = blockDim.y * gridDim.y;

    // 2d stride loop. Only executed more than once if not enough threads
    for (size_t row = y_index; row < height; row += stride_row) {
        for (size_t col = x_index; col < width; col += stride_col) {
            // T* pElement = (T*)((char*)BaseAddress + Row * pitch) + Column;
            uint8_t* this_cell = (uint8_t*)((char*)now + row * pitch_now) + col;
            uint8_t* next_cell = (uint8_t*)((char*)next + row * pitch_next) + col;
            *next_cell         = *this_cell ? 0u : 1u;
            // next[row][col] = now[row][col] ? 0u : 1u;
        }
    }
    return;
    // size_t ix = blockDim.x * blockIdx.x + threadIdx.x + 1;
    // size_t iy = blockDim.y * blockIdx.y + threadIdx.y + 1;
    // size_t id = iy * (width + 2) + ix;

    // int count = 0;
    // if (iy < height && ix < width) {
    //     count = now[id + (height + 2)] +                          // Upper neighbor
    //             now[id - (height + 2)] +                          // Lower neighbor
    //             now[id + 1] +                                     // Right neighbor
    //             now[id - 1] +                                     // Left neighbor
    //             now[id + (height + 3)] + now[id - (height + 3)] + // Diagonal neighbors
    //             now[id - (height + 1)] + now[id + (height + 1)];

    //     next[id] = (count == 3 || (count == 2 && now[id])) ? 1u : 0u;
    // }
}

void Life::updateCudaNormal()
{
    // copy host array (vector) to device
    // checkCudaErrors(hipMemcpy(d_bfr_current, m_bfr_current.data(),
    //                            sizeof(State) * m_bfr_current.size(),
    //                            hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy2D(d_bfr_current, d_current_pitch,
                                 m_bfr_current.data(), sizeof(State) * m_width,
                                 sizeof(State) * m_width, m_height,
                                 hipMemcpyHostToDevice));
    // execute kernel
    deviceOneGeneration<<<dim3{32, 32}, dim3{16, 16}>>>((uint8_t*)d_bfr_current, (uint8_t*)d_bfr_next,
                                                        d_current_pitch, d_next_pitch,
                                                        m_width, m_height);

    // copy memory back to device (blocks until kernel done)
    // checkCudaErrors(hipMemcpy(m_bfr_next.data(), d_bfr_next,
    //                            sizeof(State) * m_bfr_next.size(),
    //                            hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy2D(m_bfr_current.data(), sizeof(State) * m_width, /* dest pitch */
                                 d_bfr_next, d_next_pitch,
                                 sizeof(State) * m_width, m_height,
                                 hipMemcpyDeviceToHost));
    // new gen stored in bfr_next
    // swap the std::vectors. This only swaps the underlying pointers,
    // not the contained data. Very cheap and fast (hopefully)
    std::swap(m_bfr_current, m_bfr_next);
}
void Life::updateCudaManaged()
{
}
void Life::updateCudaPinned()
{
}

/**
 * Run the game of life rules for the specified rows
 * @param start_row first row to process
 * @param end_row last row to process
 */
inline void Life::process_chunk(size_t start_row, size_t end_row)
{
    for (size_t row = start_row; row < end_row; ++row) {
        for (size_t col = 0; col < this->m_width; ++col) {
            const auto& state = this->simulateSingleCell(row, col);
            this->setCell(row, col, state);
        }
    }
}

/**
 * Applies the Game of Life rules on a single cell, and return the next state
 *
 * @param current Read only access the the current state (for counting neighbors)
 * @param row pos of the cell
 * @param col pos of the cell
 */
inline Life::State Life::simulateSingleCell(size_t row, size_t col) const
{
    uint neighbors = countNeighbors(row, col);

    if (this->getCell(row, col) == State::Alive) // currently alive
    {
        if (neighbors < 2 || neighbors > 3) {
            return State::Dead; // Cell dies
        }
        else {
            return State::Alive; // Continues to live
        }
    }
    else // currently dead
    {
        if (neighbors == 3) {
            return State::Alive; // Cell becomes alive
        }
        else {
            return State::Dead; // Remains dead
        }
    }
}

/**
 * Find the number of live cells around a point x,y on the grid
 *
 * @param grid world to check
 * @param row pos of the cell
 * @param col pos of the cell
 * @return number of neighbors
 */
inline int Life::countNeighbors(size_t row, size_t col) const
{
    int live_count = 0;

    const long row_s = static_cast<long>(row);
    const long col_s = static_cast<long>(col);

    // Compile-time casts required for Enum -> int conversion
    live_count += static_cast<int>(this->getCellWrap(row_s + 1, col_s - 1)); // top-left
    live_count += static_cast<int>(this->getCellWrap(row_s + 1, col_s));     // top
    live_count += static_cast<int>(this->getCellWrap(row_s + 1, col_s + 1)); // top-right

    live_count += static_cast<int>(this->getCellWrap(row_s, col_s - 1)); // left
    live_count += static_cast<int>(this->getCellWrap(row_s, col_s + 1)); // right

    live_count += static_cast<int>(this->getCellWrap(row_s - 1, col_s - 1)); // bottom-left
    live_count += static_cast<int>(this->getCellWrap(row_s - 1, col_s));     // bottom
    live_count += static_cast<int>(this->getCellWrap(row_s - 1, col_s + 1)); // bottom right

    return live_count;
}

/**
 * Convert row,col specifier to the 1D vector access
 * Reads from current state
 */
inline Life::State Life::getCell(size_t row, size_t col) const
{
    return m_bfr_current[row * m_width + col];
}

/**
 * Convert row,col specifier to the 1D vector access
 * Reads from current state. Allows negative or overflow values to wrap around.
 * This has been optimized specifically for the countNeighbors operation,
 * after checking performance counting.
 */
inline Life::State Life::getCellWrap(long row, long col) const
{
    // row = (row + m_height) % m_height;
    // col = (col + m_width) % m_width;
    // branches were faster than modulo arithmetic
    // guessing branch prediction does wonders here
    if (row < 0) [[unlikely]] {
        row = m_height;
    };
    if (col < 0) [[unlikely]] {
        col = m_width;
    };
    if (row > static_cast<long>(m_height) - 1) [[unlikely]] {
        row = 0;
    };
    if (col > static_cast<long>(m_width) - 1) [[unlikely]] {
        col = 0;
    };
    return m_bfr_current[row * m_width + col];
}

/**
 * Convert row,col specifier to the 1D vector access.
 * Changes contents of next buffer
 * */
inline void Life::setCell(size_t row, size_t col, State state)
{
    m_bfr_next[row * m_width + col] = state;
}
